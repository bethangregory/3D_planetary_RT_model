//emission_gpu.cu --- routines to copy emission object to gpu

#include "Real.hpp"
#include "emission.hpp"
#include "hip/hip_runtime_api.h"

template <int N_VOXELS>
void emission<N_VOXELS>::vector_to_device(voxel_vector & device_vec, voxel_vector & host_vec, bool transfer/*=true*/) {
  //if transfer = false vector is allocated on device but not copied
  
  //allocate the host's d_vec to point at device memory
  checkCudaErrors(
		  hipMalloc((void **) &host_vec.d_vec,
			     n_voxels*sizeof(Real))
		  );
  if (transfer)
    //copy from host to host's device pointer
    checkCudaErrors(
		    hipMemcpy(host_vec.d_vec,
			       host_vec.vec,
			       n_voxels*sizeof(Real),
			       hipMemcpyHostToDevice)
		  );
  //point the device pointer at the same location we just moved memory to
  checkCudaErrors(
		  hipMemcpy(&device_vec.vec,
			     &host_vec.d_vec,
			     sizeof(Real*),
			     hipMemcpyHostToDevice)
		  );
}

template <int N_VOXELS>
void emission<N_VOXELS>::matrix_to_device(voxel_matrix & device_mat, voxel_matrix & host_mat, bool transfer/*=true*/) {
  //if transfer = false vector is allocated on device but not copied

  //allocate the host's d_vec to point at device memory
  checkCudaErrors(
		  hipMalloc((void **) &host_mat.d_mat,
			     n_voxels*n_voxels*sizeof(Real))
		  );
  if (transfer)
    //copy from host to host's device pointer
    checkCudaErrors(
		    hipMemcpy(host_mat.d_mat,
			       host_mat.mat,
			       n_voxels*n_voxels*sizeof(Real),
			       hipMemcpyHostToDevice)
		    );
  //point the device pointer at the same location we just moved memory to
  checkCudaErrors(
		  hipMemcpy(&device_mat.mat,
			     &host_mat.d_mat,
			     sizeof(Real*),
			     hipMemcpyHostToDevice)
		  );
}

template <int N_VOXELS>
void emission<N_VOXELS>::copy_to_device_influence(emission<N_VOXELS> *device_emission) {

  vector_to_device(device_emission->species_sigma, species_sigma);

  vector_to_device(device_emission->dtau_species, dtau_species);
  vector_to_device(device_emission->dtau_absorber, dtau_absorber);

  matrix_to_device(device_emission->influence_matrix, influence_matrix, false);

  vector_to_device(device_emission->tau_species_single_scattering, tau_species_single_scattering, false);
  vector_to_device(device_emission->tau_absorber_single_scattering, tau_absorber_single_scattering, false);
  vector_to_device(device_emission->singlescat, singlescat, false);
}

template <int N_VOXELS>
void emission<N_VOXELS>::copy_to_device_brightness(emission<N_VOXELS> *device_emission) {
  vector_to_device(device_emission->log_dtau_species, log_dtau_species);
  vector_to_device(device_emission->log_dtau_absorber, log_dtau_absorber);
  vector_to_device(device_emission->log_sourcefn, log_sourcefn); 
}

template <int N_VOXELS>
void emission<N_VOXELS>::vector_to_host(voxel_vector & host_vec) {

  // host_vec already has a pointer to the device location
  // (allocated in vector_to_device)
  // copy back from there
  checkCudaErrors(
		  hipMemcpy(host_vec.vec,
			     host_vec.d_vec,
			     n_voxels*sizeof(Real),
			     hipMemcpyDeviceToHost)
		  );
}

template <int N_VOXELS>
void emission<N_VOXELS>::matrix_to_host(voxel_matrix & host_mat) {

  // host_mat already has a pointer to the device location
  // (allocated in matrix_to_device)
  // copy back from there
  checkCudaErrors(
		  hipMemcpy(host_mat.mat,
			     host_mat.d_mat,
			     n_voxels*n_voxels*sizeof(Real),
			     hipMemcpyDeviceToHost)
		  );
}

template <int N_VOXELS>
void emission<N_VOXELS>::copy_influence_to_host() {
  matrix_to_host(influence_matrix);

  vector_to_host(tau_species_single_scattering);
  vector_to_host(tau_absorber_single_scattering);
  vector_to_host(singlescat);
}
